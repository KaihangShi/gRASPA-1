#include "hip/hip_runtime.h"
//##include "data_struct.h"
#include "VDW_Coulomb.cuh"
#include <hip/hip_fp16.h>
inline void VDW_CPU(const double* FFarg, const double rr_dot, const double scaling, double* result) //Lennard-Jones 12-6
{
  double arg1 = 4.0 * FFarg[0];
  double arg2 = FFarg[1] * FFarg[1];
  double arg3 = FFarg[3]; //the third element of the 3rd dimension of the array
  double temp = (rr_dot / arg2);
  double temp3 = temp * temp * temp;
  double rri3 = 1.0 / (temp3 + 0.5 * (1.0 - scaling) * (1.0 - scaling));
  double rri6 = rri3 * rri3;
  double term = arg1 * (rri3 * (rri3 - 1.0)) - arg3;
  double dlambda_term = scaling * arg1 * (rri6 * (2.0 * rri3 - 1.0));
  result[0] = scaling * term; result[1] = scaling < 1.0 ? term + (1.0 - scaling) * dlambda_term : 0.0;
}

inline void CoulombReal_CPU(double* FFParams, const double chargeA, const double chargeB, const double r, const double scaling, double* result) //energy = -q1*q2/r
{
  double prefactor = FFParams[3];
  double alpha = FFParams[4];
  double term = chargeA * chargeB * std::erfc(alpha * r);
  result[0] = prefactor * scaling * term / r;
}

inline void PBC_CPU(double* posvec, double* Cell, double* InverseCell, int* OtherParams)
{
  switch (OtherParams[0])//cubic/cuboid
      {
      case 0:
      {
        posvec[0] = posvec[0] - static_cast<int>(posvec[0] * InverseCell[0*3+0] + ((posvec[0] >= 0.0) ? 0.5 : -0.5)) * Cell[0*3+0];
        posvec[1] = posvec[1] - static_cast<int>(posvec[1] * InverseCell[1*3+1] + ((posvec[1] >= 0.0) ? 0.5 : -0.5)) * Cell[1*3+1];
        posvec[2] = posvec[2] - static_cast<int>(posvec[2] * InverseCell[2*3+2] + ((posvec[2] >= 0.0) ? 0.5 : -0.5)) * Cell[2*3+2];
        break;
      }
      default: //regardless of shape
      {
        double s[3] = {0.0, 0.0, 0.0};
        s[0]=InverseCell[0*3+0]*posvec[0]+InverseCell[1*3+0]*posvec[1]+InverseCell[2*3+0]*posvec[2];
        s[1]=InverseCell[0*3+1]*posvec[0]+InverseCell[1*3+1]*posvec[1]+InverseCell[2*3+1]*posvec[2];
        s[2]=InverseCell[0*3+2]*posvec[0]+InverseCell[1*3+2]*posvec[1]+InverseCell[2*3+2]*posvec[2];

        s[0] -= static_cast<int>(s[0] + ((s[0] >= 0.0) ? 0.5 : -0.5));
        s[1] -= static_cast<int>(s[1] + ((s[1] >= 0.0) ? 0.5 : -0.5));
        s[2] -= static_cast<int>(s[2] + ((s[2] >= 0.0) ? 0.5 : -0.5));
        // convert from abc to xyz
        posvec[0]=Cell[0*3+0]*s[0]+Cell[1*3+0]*s[1]+Cell[2*3+0]*s[2];
        posvec[1]=Cell[0*3+1]*s[0]+Cell[1*3+1]*s[1]+Cell[2*3+1]*s[2];
        posvec[2]=Cell[0*3+2]*s[0]+Cell[1*3+2]*s[1]+Cell[2*3+2]*s[2];
        break;
      }
      }

}

double Framework_energy_CPU(Boxsize Box, Atoms* Host_System, Atoms* System, ForceField FF, Components SystemComponents)
{
  ///////////////////////////////////////////////////////
  //All variables passed here should be device pointers//
  ///////////////////////////////////////////////////////
  //Copy Adsorbate to host//
  for(size_t ijk=1; ijk < SystemComponents.Total_Components; ijk++) //Skip the first one(framework)
  {
    if(Host_System[ijk].Allocate_size != System[ijk].Allocate_size)
    {
      // if the host allocate_size is different from the device, allocate more space on the host
      Host_System[ijk].x         = (double*) malloc(System[ijk].Allocate_size*sizeof(double));
      Host_System[ijk].y         = (double*) malloc(System[ijk].Allocate_size*sizeof(double));
      Host_System[ijk].z         = (double*) malloc(System[ijk].Allocate_size*sizeof(double));
      Host_System[ijk].scale     = (double*) malloc(System[ijk].Allocate_size*sizeof(double));
      Host_System[ijk].charge    = (double*) malloc(System[ijk].Allocate_size*sizeof(double));
      Host_System[ijk].scaleCoul = (double*) malloc(System[ijk].Allocate_size*sizeof(double));
      Host_System[ijk].Type      = (size_t*) malloc(System[ijk].Allocate_size*sizeof(size_t));
      Host_System[ijk].MolID     = (size_t*) malloc(System[ijk].Allocate_size*sizeof(size_t));
      Host_System[ijk].size      = System[ijk].size; 
      Host_System[ijk].Allocate_size = System[ijk].Allocate_size;
    }
  
    if(Host_System[ijk].Allocate_size = System[ijk].Allocate_size) //means there is no more space allocated on the device than host, otherwise, allocate more on host
    {
      hipMemcpy(Host_System[ijk].x, System[ijk].x, sizeof(double)*System[ijk].Allocate_size, hipMemcpyDeviceToHost);
      hipMemcpy(Host_System[ijk].y, System[ijk].y, sizeof(double)*System[ijk].Allocate_size, hipMemcpyDeviceToHost);
      hipMemcpy(Host_System[ijk].z, System[ijk].z, sizeof(double)*System[ijk].Allocate_size, hipMemcpyDeviceToHost);
      hipMemcpy(Host_System[ijk].scale, System[ijk].scale, sizeof(double)*System[ijk].Allocate_size, hipMemcpyDeviceToHost);
      hipMemcpy(Host_System[ijk].charge, System[ijk].charge, sizeof(double)*System[ijk].Allocate_size, hipMemcpyDeviceToHost);
      hipMemcpy(Host_System[ijk].scaleCoul, System[ijk].scaleCoul, sizeof(double)*System[ijk].Allocate_size, hipMemcpyDeviceToHost);
      hipMemcpy(Host_System[ijk].Type, System[ijk].Type, sizeof(size_t)*System[ijk].Allocate_size, hipMemcpyDeviceToHost);
      hipMemcpy(Host_System[ijk].MolID, System[ijk].MolID, sizeof(size_t)*System[ijk].Allocate_size, hipMemcpyDeviceToHost);
      Host_System[ijk].size = System[ijk].size;
      //printf("CPU CHECK: comp: %zu, Host Allocate_size: %zu, Allocate_size: %zu\n", ijk, Host_System[ijk].Allocate_size, System[ijk].Allocate_size);
    }
  }
  double Total_energy = 0.0; size_t count = 0; size_t cutoff_count=0;
  for(size_t compi=0; compi < SystemComponents.Total_Components; compi++) 
  {
    const Atoms Component=Host_System[compi];
    //printf("compi: %zu, size: %zu\n", compi, Component.size);
    for(size_t i=0; i<Component.size; i++)
    {
      //printf("comp: %zu, i: %zu, x: %.10f\n", compi, i, Component.x[i]);
      const double scaleA = Component.scale[i];
      const double chargeA = Component.charge[i];
      const double scalingCoulombA = Component.scaleCoul[i];
      const size_t typeA = Component.Type[i];
      const size_t MoleculeID = Component.MolID[i];
      for(size_t compj=0; compj < SystemComponents.Total_Components; compj++)
      {
        if(!((compi == 0) && (compj == 0))) //ignore fraemwrok-framework interaction
        {
          const Atoms Componentj=Host_System[compj];
          for(size_t j=0; j<Componentj.size; j++)
          {
            const double scaleB = Componentj.scale[j];
            const double chargeB = Componentj.charge[j];
            const double scalingCoulombB = Componentj.scaleCoul[j];
            const size_t typeB = Componentj.Type[j];
            const size_t MoleculeIDB = Componentj.MolID[j];
            if(!((MoleculeID == MoleculeIDB) &&(compi == compj)))
            {
              count++;
              double posvec[3] = {Component.x[i] - Componentj.x[j], Component.y[i] - Componentj.y[j], Component.z[i] - Componentj.z[j]};
              PBC_CPU(posvec, Box.Cell, Box.InverseCell, FF.OtherParams);
              const double rr_dot = posvec[0]*posvec[0] + posvec[1]*posvec[1] + posvec[2]*posvec[2];
              //printf("i: %zu, j: %zu, rr_dot: %.10f\n", i,j,rr_dot);
              double result[2] = {0.0, 0.0};
              if(rr_dot < FF.FFParams[1])
              {
                cutoff_count++;
                const double scaling = scaleA * scaleB;
                const size_t row = typeA*FF.size+typeB;
                const double FFarg[4] = {FF.epsilon[row], FF.sigma[row], FF.z[row], FF.shift[row]};
                VDW_CPU(FFarg, rr_dot, scaling, result);
                Total_energy += 0.5*result[0];
              }
              if (!FF.noCharges && rr_dot < FF.FFParams[2])
              {
                const double r = sqrt(rr_dot);
                const double scalingCoul = scalingCoulombA * scalingCoulombB;
                double resultCoul[2] = {0.0, 0.0};
                CoulombReal_CPU(FF.FFParams, chargeA, chargeB, r, scalingCoul, resultCoul);
                Total_energy += 0.5*resultCoul[0]; //prefactor merged in the CoulombReal function
              }
            }
          }
        }
      }
    }  
  }
  //printf("%zu interactions, within cutoff: %zu, energy: %.10f\n", count, Total_energy, cutoff_count);
  return Total_energy;
}

////////////////////////////// GPU CODE //////////////////////////

__device__ void VDW(const double* FFarg, const double rr_dot, const double scaling, double* result) //Lennard-Jones 12-6
{
  double arg1 = 4.0 * FFarg[0];
  double arg2 = FFarg[1] * FFarg[1];
  double arg3 = FFarg[3]; //the third element of the 3rd dimension of the array
  double temp = (rr_dot / arg2);
  double temp3 = temp * temp * temp;
  double rri3 = 1.0 / (temp3 + 0.5 * (1.0 - scaling) * (1.0 - scaling));
  double rri6 = rri3 * rri3;
  double term = arg1 * (rri3 * (rri3 - 1.0)) - arg3;
  double dlambda_term = scaling * arg1 * (rri6 * (2.0 * rri3 - 1.0));
  result[0] = scaling * term; result[1] = scaling < 1.0 ? term + (1.0 - scaling) * dlambda_term : 0.0;
}

__device__ void CoulombReal(double* FFParams, const double chargeA, const double chargeB, const double r, const double scaling, double* result) //energy = -q1*q2/r
{
  double prefactor = FFParams[3];
  double alpha = FFParams[4];
  double term = chargeA * chargeB * std::erfc(alpha * r);
  result[0] = prefactor * scaling * term / r;
}

__device__ void PBC(double* posvec, double* Cell, double* InverseCell, int* OtherParams)
{
  switch (OtherParams[0])//cubic/cuboid
      {
      case 0:
      {
        posvec[0] = posvec[0] - static_cast<int>(posvec[0] * InverseCell[0*3+0] + ((posvec[0] >= 0.0) ? 0.5 : -0.5)) * Cell[0*3+0];
        posvec[1] = posvec[1] - static_cast<int>(posvec[1] * InverseCell[1*3+1] + ((posvec[1] >= 0.0) ? 0.5 : -0.5)) * Cell[1*3+1];
        posvec[2] = posvec[2] - static_cast<int>(posvec[2] * InverseCell[2*3+2] + ((posvec[2] >= 0.0) ? 0.5 : -0.5)) * Cell[2*3+2];
        break;
      }
      default: //regardless of shape
      {
        double s[3] = {0.0, 0.0, 0.0};
        s[0]=InverseCell[0*3+0]*posvec[0]+InverseCell[1*3+0]*posvec[1]+InverseCell[2*3+0]*posvec[2];
        s[1]=InverseCell[0*3+1]*posvec[0]+InverseCell[1*3+1]*posvec[1]+InverseCell[2*3+1]*posvec[2];
        s[2]=InverseCell[0*3+2]*posvec[0]+InverseCell[1*3+2]*posvec[1]+InverseCell[2*3+2]*posvec[2];

        s[0] -= static_cast<int>(s[0] + ((s[0] >= 0.0) ? 0.5 : -0.5));
        s[1] -= static_cast<int>(s[1] + ((s[1] >= 0.0) ? 0.5 : -0.5));
        s[2] -= static_cast<int>(s[2] + ((s[2] >= 0.0) ? 0.5 : -0.5));
        // convert from abc to xyz
        posvec[0]=Cell[0*3+0]*s[0]+Cell[1*3+0]*s[1]+Cell[2*3+0]*s[2];
        posvec[1]=Cell[0*3+1]*s[0]+Cell[1*3+1]*s[1]+Cell[2*3+1]*s[2];
        posvec[2]=Cell[0*3+2]*s[0]+Cell[1*3+2]*s[1]+Cell[2*3+2]*s[2];
        break;
      }
      }

}

__device__ void VDW_float(const float* FFarg, const float rr_dot, const float scaling, float* result) //Lennard-Jones 12-6
{
  float arg1 = 4.0 * FFarg[0];
  float arg2 = FFarg[1] * FFarg[1];
  float arg3 = FFarg[3]; //the third element of the 3rd dimension of the array
  float temp = (rr_dot / arg2);
  float temp3 = temp * temp * temp;
  float rri3 = 1.0 / (temp3 + 0.5 * (1.0 - scaling) * (1.0 - scaling));
  float rri6 = rri3 * rri3;
  float term = arg1 * (rri3 * (rri3 - 1.0)) - arg3;
  float dlambda_term = scaling * arg1 * (rri6 * (2.0 * rri3 - 1.0));
  result[0] = scaling * term; result[1] = scaling < 1.0 ? term + (1.0 - scaling) * dlambda_term : 0.0;
}

__device__ void CoulombReal_float(double* FFParams, const float chargeA, const float chargeB, const float r, const float scaling, float* result) //energy = -q1*q2/r
{
  float prefactor = __double2float_rd(FFParams[3]);
  float alpha     = __double2float_rd(FFParams[4]);
  float term = chargeA * chargeB * std::erfc(alpha * r);
  result[0] = prefactor * scaling * term / r;
}

__device__ void PBC_float(float* posvec, float* Cell, float* InverseCell, int* OtherParams)
{
  switch (OtherParams[0])//cubic/cuboid
  {
    case 0:
    {
      posvec[0] = posvec[0] - static_cast<int>(posvec[0] * InverseCell[0*3+0] + ((posvec[0] >= 0.0) ? 0.5 : -0.5)) * Cell[0*3+0];
      posvec[1] = posvec[1] - static_cast<int>(posvec[1] * InverseCell[1*3+1] + ((posvec[1] >= 0.0) ? 0.5 : -0.5)) * Cell[1*3+1];
      posvec[2] = posvec[2] - static_cast<int>(posvec[2] * InverseCell[2*3+2] + ((posvec[2] >= 0.0) ? 0.5 : -0.5)) * Cell[2*3+2];
      break;
    }
    default: //regardless of shape
    {
      float s[3] = {0.0, 0.0, 0.0};
      s[0]=InverseCell[0*3+0]*posvec[0]+InverseCell[1*3+0]*posvec[1]+InverseCell[2*3+0]*posvec[2];
      s[1]=InverseCell[0*3+1]*posvec[0]+InverseCell[1*3+1]*posvec[1]+InverseCell[2*3+1]*posvec[2];
      s[2]=InverseCell[0*3+2]*posvec[0]+InverseCell[1*3+2]*posvec[1]+InverseCell[2*3+2]*posvec[2];
      s[0] -= static_cast<int>(s[0] + ((s[0] >= 0.0) ? 0.5 : -0.5));
      s[1] -= static_cast<int>(s[1] + ((s[1] >= 0.0) ? 0.5 : -0.5));
      s[2] -= static_cast<int>(s[2] + ((s[2] >= 0.0) ? 0.5 : -0.5));
      // convert from abc to xyz
      posvec[0]=Cell[0*3+0]*s[0]+Cell[1*3+0]*s[1]+Cell[2*3+0]*s[2];
      posvec[1]=Cell[0*3+1]*s[0]+Cell[1*3+1]*s[1]+Cell[2*3+1]*s[2];
      posvec[2]=Cell[0*3+2]*s[0]+Cell[1*3+2]*s[1]+Cell[2*3+2]*s[2];
      break;
    }
  }
}


__global__ void one_thread_GPU_test(Boxsize Box, Atoms* System, ForceField FF, double* xxx)
{
  bool DEBUG=false;
  //Zhao's note: added temp_xxx values for checking individual energy for each molecule//
  double temp_energy = 0.0; double temp_firstbead = 0.0; double temp_chain = 0.0; size_t temp_i = 0; int temp_count = -1;
  double Total_energy = 0.0; size_t count = 0; size_t cutoff_count=0;
  for(size_t compi=0; compi < 2; compi++) //Zhao's note: hard coded component, change
  {
    const Atoms Component=System[compi];
    //printf("GPU CHECK: Comp: %lu, Comp size: %lu, Allocate size: %lu\n", compi, Component.size, Component.Allocate_size);
    for(size_t i=0; i<Component.size; i++)
    {
      //printf("comp: %lu, i: %lu, x: %.10f\n", compi, i, Component.x[i]);
      const double scaleA = Component.scale[i];
      const double chargeA = Component.charge[i];
      const double scalingCoulombA = Component.scaleCoul[i];
      const size_t typeA = Component.Type[i];
      const size_t MoleculeID = Component.MolID[i];
      if(DEBUG){if(MoleculeID == 5) temp_count++;} //For testing individual molecule energy//
      for(size_t compj=0; compj < 2; compj++) //Zhao's note: hard coded component, change
      {
        if(!((compi == 0) && (compj == 0))) //ignore fraemwrok-framework interaction
        {
          const Atoms Componentj=System[compj];
          for(size_t j=0; j<Componentj.size; j++)
          {
            const double scaleB = Componentj.scale[j];
            const double chargeB = Componentj.charge[j];
            const double scalingCoulombB = Componentj.scaleCoul[j];
            const size_t typeB = Componentj.Type[j];
            const size_t MoleculeIDB = Componentj.MolID[j];
            if(!((MoleculeID == MoleculeIDB) &&(compi == compj)))
            {
              count++;
              double posvec[3] = {Component.x[i] - Componentj.x[j], Component.y[i] - Componentj.y[j], Component.z[i] - Componentj.z[j]};
              PBC(posvec, Box.Cell, Box.InverseCell, FF.OtherParams);
              const double rr_dot = posvec[0]*posvec[0] + posvec[1]*posvec[1] + posvec[2]*posvec[2];
              double result[2] = {0.0, 0.0};
              if(rr_dot < FF.FFParams[1])
              {
                cutoff_count++;
                const double scaling = scaleA * scaleB;
                const size_t row = typeA*FF.size+typeB;
                const double FFarg[4] = {FF.epsilon[row], FF.sigma[row], FF.z[row], FF.shift[row]};
                VDW(FFarg, rr_dot, scaling, result);
                Total_energy += 0.5*result[0];
                if(DEBUG){if(MoleculeID == 5)
                { 
                  temp_energy += result[0];
                  if(temp_count == 0){temp_firstbead += result[0];}
                  else {temp_chain += result[0];}
                } 
              }}
              //  printf("SPECIEL CHECK: compi: %lu, i: %lu, compj: %lu, j: %lu, pos: %.5f, %.5f, %.5f, rr_dot: %.10f, energy: %.10f\n", compi,i,compj,j,Component.x[i], Component.y[i], Component.z[i], rr_dot, result[0]);
              if (!FF.noCharges && rr_dot < FF.FFParams[2])
              {
                const double r = sqrt(rr_dot);
                const double scalingCoul = scalingCoulombA * scalingCoulombB;
                double resultCoul[2] = {0.0, 0.0};
                CoulombReal(FF.FFParams, chargeA, chargeB, r, scalingCoul, resultCoul);
                Total_energy += 0.5*resultCoul[0]; //prefactor merged in the CoulombReal function
              }
            }
          }
        }
      }
    }
  }
  if(DEBUG) printf("For Molecule 5, energy: %.10f, firstbead: %.10f, chain: %.10f\n", temp_energy, temp_firstbead, temp_chain);
  xxx[0] = Total_energy;
  //printf("xxx: %.10f\n", Total_energy);
}
__global__ void Framework_energy_difference_SoA(Boxsize Box, Atoms* System, Atoms Mol, Atoms NewMol, ForceField FF, double* y, double* dUdlambda, size_t ComponentID, size_t totalthreads) // Consider to change this for polyatomic
{
  ///////////////////////////////////////////////////////
  //All variables passed here should be device pointers//
  ///////////////////////////////////////////////////////
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < totalthreads)
  {
  y[i] = 0.0; dUdlambda[i] = 0.0;
  size_t comp = 0;
  const size_t NumberComp = 2; //Need to change this for multi-component
  size_t posi = i; size_t totalsize= 0;
  //printf("%lu, %lu\n", System[0].size, System[1].size);
  for(size_t ijk = 0; ijk < NumberComp; ijk++)
  {
    totalsize += System[ijk].size;
    if(posi >= totalsize)
    {
      comp++;
      posi -= System[ijk].size;
    }
  }
  //printf("thread: %lu, comp: %lu, posi: %lu\n", i,comp, posi);

  const Atoms Component=System[comp];
  const double scaleA = Component.scale[posi];
  const double chargeA = Component.charge[posi];
  const double scalingCoulombA = Component.scaleCoul[posi];
  const size_t typeA = Component.Type[posi];
  const size_t MoleculeID = Component.MolID[posi];
  double tempy = 0.0; double tempdU = 0.0;
  if(!((MoleculeID == NewMol.MolID[0]) &&(comp == ComponentID))) //ComponentID: Component ID for the molecule being translated
  {
  for (size_t j = 0; j < NewMol.size; j++) //NEW POSITION
  {
    //printf("i: %lu, posi: %lu, MoleculeID: %lu, NewMol.MolID[0]: %lu, ComponentID: %lu, x: %.10f\n", i, posi, MoleculeID, NewMol.MolID[0], ComponentID, Component.x[posi]);
    double posvec[3] = {Component.x[posi] - NewMol.x[j], Component.y[posi] - NewMol.y[j], Component.z[posi] - NewMol.z[j]};

    PBC(posvec, Box.Cell, Box.InverseCell, FF.OtherParams);
    const double rr_dot = posvec[0]*posvec[0] + posvec[1]*posvec[1] + posvec[2]*posvec[2];
    if(rr_dot < FF.FFParams[1])
    {
      double result[2] = {0.0, 0.0};
      const size_t typeB = NewMol.Type[j];
      const double scaleB = NewMol.scale[j];
      const double scaling = scaleA * scaleB;
      const size_t row = typeA*FF.size+typeB;
      const double FFarg[4] = {FF.epsilon[row], FF.sigma[row], FF.z[row], FF.shift[row]};
      VDW(FFarg, rr_dot, scaling, result);
      tempy += result[0];
      tempdU += result[1];
    }
    //printf("comp: %lu, i: %lu, j: %lu, pos: %.5f, Newpos: %.5f, rr_dot: %.10f, energy: %.10f\n", comp, posi, NewMol.MolID[0],Component.x[posi], NewMol.x[j], rr_dot, tempy);
  
    if (!FF.noCharges && rr_dot < FF.FFParams[2])
    {
      const double chargeB = NewMol.charge[j];
      const double scalingCoulombB = NewMol.scaleCoul[j];
      const double r = sqrt(rr_dot);
      const double scalingCoul = scalingCoulombA * scalingCoulombB;
      double resultCoul[2] = {0.0, 0.0};
      CoulombReal(FF.FFParams, chargeA, chargeB, r, scalingCoul, resultCoul);
      tempy += resultCoul[0]; //prefactor merged in the CoulombReal function
    }
  }
  for (size_t j = 0; j < Mol.size; j++) //OLD POSITION
  {
    double posvec[3] = {Component.x[posi] - Mol.x[j], Component.y[posi] - Mol.y[j], Component.z[posi] - Mol.z[j]};
    PBC(posvec, Box.Cell, Box.InverseCell, FF.OtherParams);
    const double rr_dot = posvec[0]*posvec[0] + posvec[1]*posvec[1] + posvec[2]*posvec[2];
    //printf("old, %lu, %.10f, x: %.10f, y: %.10f, z: %.10f, Molx: %.10f, Moly: %.10f, Molz: %.10f\n", i, rr_dot, Component.x[i], Component.y[i], Component.z[i], Mol.x[j], Mol.y[j], Mol.z[j]);
    double result[2] = {0.0, 0.0};
    if(rr_dot < FF.FFParams[1])
    {
      const size_t typeB = Mol.Type[j];
      const double scaleB = Mol.scale[j];
      const double scaling = scaleA * scaleB;
      const size_t row = typeA*FF.size+typeB;
      const double FFarg[4] = {FF.epsilon[row], FF.sigma[row], FF.z[row], FF.shift[row]};
      VDW(FFarg, rr_dot, scaling, result);
      tempy -= result[0];
      tempdU -= result[1];
    }
    //printf("comp: %lu, i: %lu, j: %lu, pos: %.5f, Oldpos: %.5f, rr_dot: %.10f, energy: %.10f, y[i]: %.10f\n", comp, posi, NewMol.MolID[0],Component.x[posi], Mol.x[j], rr_dot, result[0], y[i]);
    if (!FF.noCharges && rr_dot < FF.FFParams[2])
    {
      const double chargeB = Mol.charge[j];
      const double scalingCoulombB = Mol.scaleCoul[j];
      const double r = sqrt(rr_dot);
      const double scalingCoul = scalingCoulombA * scalingCoulombB;
      double resultCoul[2] = {0.0, 0.0};
      CoulombReal(FF.FFParams, chargeA, chargeB, r, scalingCoul, resultCoul);
      tempy -= resultCoul[0]; //prefactor merged in the CoulombReal function
    }
  }
  y[i] = tempy; dUdlambda[i] = tempdU;
  }
  }
}

__global__ void Collapse_Framework_Energy(Boxsize Box, Atoms* System, Atoms NewMol, ForceField FF, double* y, size_t ComponentID, size_t totalAtoms, size_t totalthreads, size_t trialsize)
{
  // TEST THE SPEED OF THIS //
  // CHANGED HOW THE ith element of framework positions and y are written/accessed //
  // Zhao's note: added trialsize to distinguish single-bead and trial orientations //
  ///////////////////////////////////////////////////////
  //All variables passed here should be device pointers//
  ///////////////////////////////////////////////////////
  size_t ij = blockIdx.x * blockDim.x + threadIdx.x;
  if(ij < totalthreads)
  {
  const double VDWCutoff = FF.FFParams[1];
  const double CoulCutoff = FF.FFParams[2];
  const bool   noCharges = FF.noCharges;
  // Manually fusing/collapsing the loop //
  //size_t i = ij/Mol.size; size_t j = ij%Mol.size;
  size_t i = ij%totalAtoms; size_t j = ij/totalAtoms;
  //Zhao's note: use trialsize for multiple atoms in one trial orientation//
  size_t k = j/trialsize;
  size_t comp = 0;
  const size_t NumberComp = 2; //Zhao's note: need to change for multicomponent
  size_t posi = i; size_t totalsize= 0;
  //printf("%lu, %lu\n", System[0].size, System[1].size);
  for(size_t ijk = 0; ijk < NumberComp; ijk++)
  {
    totalsize += System[ijk].size;
    if(posi >= totalsize)
    {
      comp++;
      posi -= System[ijk].size;
    }
  }
  //printf("thread: %lu, comp: %lu, posi: %lu\n", i,comp, posi);

  const Atoms Component=System[comp];
  const double scaleA = Component.scale[posi];
  const double chargeA = Component.charge[posi];
  const double scalingCoulombA = Component.scaleCoul[posi];
  const size_t typeA = Component.Type[posi];
  const size_t MoleculeID = Component.MolID[posi];

  double Pos[3] = {Component.x[posi], Component.y[posi], Component.z[posi]};
  double tempy = 0.0;
  if(!((MoleculeID == NewMol.MolID[0]) &&(comp == ComponentID)))
  {
    double posvec[3] = {Pos[0] - NewMol.x[j], Pos[1] - NewMol.y[j], Pos[2] - NewMol.z[j]};

    PBC(posvec, Box.Cell, Box.InverseCell, FF.OtherParams);
    const double rr_dot = posvec[0]*posvec[0] + posvec[1]*posvec[1] + posvec[2]*posvec[2];
    if(rr_dot < VDWCutoff)
    {
      double result[2] = {0.0, 0.0};
      const size_t typeB = NewMol.Type[j];
      const double scaleB = NewMol.scale[j];
      const double scaling = scaleA * scaleB;
      const size_t row = typeA*FF.size+typeB;
      const double FFarg[4] = {FF.epsilon[row], FF.sigma[row], FF.z[row], FF.shift[row]};
      VDW(FFarg, rr_dot, scaling, result);
      tempy += result[0];
    }

    if (!noCharges && rr_dot < CoulCutoff)
    {
      const double chargeB = NewMol.charge[j];
      const double scalingCoulombB = NewMol.scaleCoul[j];
      const double r = sqrt(rr_dot);
      const double scalingCoul = scalingCoulombA * scalingCoulombB;
      double resultCoul[2] = {0.0, 0.0};
      CoulombReal(FF.FFParams, chargeA, chargeB, r, scalingCoul, resultCoul);
      tempy += resultCoul[0]; //prefactor merged in the CoulombReal function
    }
  }
  y[ij] = tempy;
  //printf("ij:%lu, i: %lu, j:%lu, E: %.10f\n",ij, i,j, tempy);
  }
}

__global__ void Collapse_Framework_Energy_OVERLAP_FLOAT(Boxsize Box, Atoms* System, Atoms NewMol, ForceField FF, double* y, size_t ComponentID, size_t totalAtoms, bool* flag, size_t totalthreads, size_t trialsize, float* y_float)
{
  // TEST THE SPEED OF THIS //
  // CHANGED HOW THE ith element of framework positions and y are written/accessed //
  ///////////////////////////////////////////////////////
  //All variables passed here should be device pointers//
  ///////////////////////////////////////////////////////
  size_t ij = blockIdx.x * blockDim.x + threadIdx.x;
  if(ij < totalthreads)
  {
  const double OverlapCriteria = FF.FFParams[0];
  const double VDWCutoff = FF.FFParams[1];
  const double CoulCutoff = FF.FFParams[2];
  const bool   noCharges = FF.noCharges;
  y[ij] = 0.0;
  // Manually fusing/collapsing the loop //
  //size_t i = ij/Mol.size; size_t j = ij%Mol.size;
  size_t i = ij%totalAtoms; size_t j = ij/totalAtoms;
  //Zhao's note: use trialsize for multiple atoms in one trial orientation//
  size_t k = j/trialsize;
  //printf("ij: %lu, i: %lu, j: %lu, trial: %lu, totalAtoms: %lu, totalthreads: %lu\n", ij,i,j,k,totalAtoms, totalthreads);
  size_t comp = 0;
  const size_t NumberComp = 2; //Zhao's note: need to change here for multicomponent
  size_t posi = i; size_t totalsize= 0;
  //printf("%lu, %lu\n", System[0].size, System[1].size);
  for(size_t ijk = 0; ijk < NumberComp; ijk++)
  {
    totalsize += System[ijk].size;
    if(posi >= totalsize)
    {
      comp++;
      posi -= System[ijk].size;
    }
  }

  const Atoms Component=System[comp];
  const double scaleA = Component.scale[posi];
  const double chargeA = Component.charge[posi];
  const double scalingCoulombA = Component.scaleCoul[posi];
  const size_t typeA = Component.Type[posi];
  const size_t MoleculeID = Component.MolID[posi];

  double Pos[3] = {Component.x[posi], Component.y[posi], Component.z[posi]};
  double tempy = 0.0;
  //if(j == 6) printf("PAIR CHECK: i: %lu, j: %lu, MoleculeID: %lu, NewMol.MolID: %lu\n", i,j,MoleculeID, NewMol.MolID[0]);
  if(!((MoleculeID == NewMol.MolID[0]) &&(comp == ComponentID)))
  {
    double posvec[3] = {Pos[0] - NewMol.x[j], Pos[1] - NewMol.y[j], Pos[2] - NewMol.z[j]};
    //printf("thread: %lu, i:%lu, j:%lu, comp: %lu, posi: %lu\n", ij,i,j,comp, posi);

    PBC(posvec, Box.Cell, Box.InverseCell, FF.OtherParams);
    const double rr_dot = posvec[0]*posvec[0] + posvec[1]*posvec[1] + posvec[2]*posvec[2];
    if(rr_dot < VDWCutoff)
    {
      double result[2] = {0.0, 0.0};
      const size_t typeB = NewMol.Type[j];
      const double scaleB = NewMol.scale[j];
      const double scaling = scaleA * scaleB;
      const size_t row = typeA*FF.size+typeB;
      const double FFarg[4] = {FF.epsilon[row], FF.sigma[row], FF.z[row], FF.shift[row]};
      VDW(FFarg, rr_dot, scaling, result); 
      if(result[0] > OverlapCriteria){ flag[k]=true;}
      tempy += result[0];
    }

    if (!noCharges && rr_dot < CoulCutoff)
    {
      const double chargeB = NewMol.charge[j];
      const double scalingCoulombB = NewMol.scaleCoul[j];
      const double r = sqrt(rr_dot);
      const double scalingCoul = scalingCoulombA * scalingCoulombB;
      double resultCoul[2] = {0.0, 0.0};
      CoulombReal(FF.FFParams, chargeA, chargeB, r, scalingCoul, resultCoul);
      tempy += resultCoul[0]; //prefactor merged in the CoulombReal function
    }
  }
  y[ij] = tempy;
  y_float[ij] = __double2float_rd(tempy);
  }
}

__global__ void Collapse_Framework_Energy_OVERLAP_HALF(Boxsize Box, Atoms* System, Atoms NewMol, ForceField FF, double* y, size_t ComponentID, size_t totalAtoms, bool* flag, size_t totalthreads, size_t trialsize, half* y_half)
{
  // TEST THE SPEED OF THIS //
  // CHANGED HOW THE ith element of framework positions and y are written/accessed //
  ///////////////////////////////////////////////////////
  //All variables passed here should be device pointers//
  ///////////////////////////////////////////////////////
  size_t ij = blockIdx.x * blockDim.x + threadIdx.x;
  if(ij < totalthreads)
  {
  const double OverlapCriteria = FF.FFParams[0];
  const double VDWCutoff = FF.FFParams[1];
  const double CoulCutoff = FF.FFParams[2];
  const bool   noCharges = FF.noCharges;
  y[ij] = 0.0;
  // Manually fusing/collapsing the loop //
  //size_t i = ij/Mol.size; size_t j = ij%Mol.size;
  size_t i = ij%totalAtoms; size_t j = ij/totalAtoms;
  //Zhao's note: use trialsize for multiple atoms in one trial orientation//
  size_t k = j/trialsize;
  //printf("ij: %lu, i: %lu, j: %lu, trial: %lu, totalAtoms: %lu, totalthreads: %lu\n", ij,i,j,k,totalAtoms, totalthreads);
  size_t comp = 0;
  const size_t NumberComp = 2; //Zhao's note: need to change here for multicomponent
  size_t posi = i; size_t totalsize= 0;
  //printf("%lu, %lu\n", System[0].size, System[1].size);
  for(size_t ijk = 0; ijk < NumberComp; ijk++)
  {
    totalsize += System[ijk].size;
    if(posi >= totalsize)
    {
      comp++;
      posi -= System[ijk].size;
    }
  }

  const Atoms Component=System[comp];
  const double scaleA = Component.scale[posi];
  const double chargeA = Component.charge[posi];
  const double scalingCoulombA = Component.scaleCoul[posi];
  const size_t typeA = Component.Type[posi];
  const size_t MoleculeID = Component.MolID[posi];

  double Pos[3] = {Component.x[posi], Component.y[posi], Component.z[posi]};
  double tempy = 0.0;
  //if(j == 6) printf("PAIR CHECK: i: %lu, j: %lu, MoleculeID: %lu, NewMol.MolID: %lu\n", i,j,MoleculeID, NewMol.MolID[0]);
  if(!((MoleculeID == NewMol.MolID[0]) &&(comp == ComponentID)))
  {
    double posvec[3] = {Pos[0] - NewMol.x[j], Pos[1] - NewMol.y[j], Pos[2] - NewMol.z[j]};
    //printf("thread: %lu, i:%lu, j:%lu, comp: %lu, posi: %lu\n", ij,i,j,comp, posi);

    PBC(posvec, Box.Cell, Box.InverseCell, FF.OtherParams);
    const double rr_dot = posvec[0]*posvec[0] + posvec[1]*posvec[1] + posvec[2]*posvec[2];
    if(rr_dot < VDWCutoff)
    {
      double result[2] = {0.0, 0.0};
      const size_t typeB = NewMol.Type[j];
      const double scaleB = NewMol.scale[j];
      const double scaling = scaleA * scaleB;
      const size_t row = typeA*FF.size+typeB;
      const double FFarg[4] = {FF.epsilon[row], FF.sigma[row], FF.z[row], FF.shift[row]};
      VDW(FFarg, rr_dot, scaling, result); 
      if(result[0] > OverlapCriteria){ flag[k]=true;}
      tempy += result[0];
    }

    if (!noCharges && rr_dot < CoulCutoff)
    {
      const double chargeB = NewMol.charge[j];
      const double scalingCoulombB = NewMol.scaleCoul[j];
      const double r = sqrt(rr_dot);
      const double scalingCoul = scalingCoulombA * scalingCoulombB;
      double resultCoul[2] = {0.0, 0.0};
      CoulombReal(FF.FFParams, chargeA, chargeB, r, scalingCoul, resultCoul);
      tempy += resultCoul[0]; //prefactor merged in the CoulombReal function
    }
  }
  y[ij] = tempy;
  y_half[ij] = __double2half(tempy);
  }
}

__global__ void Collapse_Framework_Energy_OVERLAP(Boxsize Box, Atoms* System, Atoms NewMol, ForceField FF, double* y, size_t ComponentID, size_t totalAtoms, bool* flag, size_t totalthreads, size_t trialsize)
{
  // TEST THE SPEED OF THIS //
  // CHANGED HOW THE ith element of framework positions and y are written/accessed //
  ///////////////////////////////////////////////////////
  //All variables passed here should be device pointers//
  ///////////////////////////////////////////////////////
  size_t ij = blockIdx.x * blockDim.x + threadIdx.x;
  if(ij < totalthreads)
  {
  const double OverlapCriteria = FF.FFParams[0];
  const double VDWCutoff = FF.FFParams[1];
  const double CoulCutoff = FF.FFParams[2];
  const bool   noCharges = FF.noCharges;
  y[ij] = 0.0;
  // Manually fusing/collapsing the loop //
  //size_t i = ij/Mol.size; size_t j = ij%Mol.size;
  size_t i = ij%totalAtoms; size_t j = ij/totalAtoms;
  //Zhao's note: use trialsize for multiple atoms in one trial orientation//
  size_t k = j/trialsize;
  //printf("ij: %lu, i: %lu, j: %lu, trial: %lu, totalAtoms: %lu, totalthreads: %lu\n", ij,i,j,k,totalAtoms, totalthreads);
  size_t comp = 0;
  const size_t NumberComp = 2; //Zhao's note: need to change here for multicomponent
  size_t posi = i; size_t totalsize= 0;
  //printf("%lu, %lu\n", System[0].size, System[1].size);
  for(size_t ijk = 0; ijk < NumberComp; ijk++)
  {
    totalsize += System[ijk].size;
    if(posi >= totalsize)
    {
      comp++;
      posi -= System[ijk].size;
    }
  }

  const Atoms Component=System[comp];
  const double scaleA = Component.scale[posi];
  const double chargeA = Component.charge[posi];
  const double scalingCoulombA = Component.scaleCoul[posi];
  const size_t typeA = Component.Type[posi];
  const size_t MoleculeID = Component.MolID[posi];

  double Pos[3] = {Component.x[posi], Component.y[posi], Component.z[posi]};
  double tempy = 0.0;
  //if(j == 6) printf("PAIR CHECK: i: %lu, j: %lu, MoleculeID: %lu, NewMol.MolID: %lu\n", i,j,MoleculeID, NewMol.MolID[0]);
  if(!((MoleculeID == NewMol.MolID[0]) &&(comp == ComponentID)))
  {
    double posvec[3] = {Pos[0] - NewMol.x[j], Pos[1] - NewMol.y[j], Pos[2] - NewMol.z[j]};
    //printf("thread: %lu, i:%lu, j:%lu, comp: %lu, posi: %lu\n", ij,i,j,comp, posi);

    PBC(posvec, Box.Cell, Box.InverseCell, FF.OtherParams);
    const double rr_dot = posvec[0]*posvec[0] + posvec[1]*posvec[1] + posvec[2]*posvec[2];
    if(rr_dot < VDWCutoff)
    {
      double result[2] = {0.0, 0.0};
      const size_t typeB = NewMol.Type[j];
      const double scaleB = NewMol.scale[j];
      const double scaling = scaleA * scaleB;
      const size_t row = typeA*FF.size+typeB;
      const double FFarg[4] = {FF.epsilon[row], FF.sigma[row], FF.z[row], FF.shift[row]};
      VDW(FFarg, rr_dot, scaling, result); 
      if(result[0] > OverlapCriteria){ flag[k]=true;}
      tempy += result[0];
    }

    if (!noCharges && rr_dot < CoulCutoff)
    {
      const double chargeB = NewMol.charge[j];
      const double scalingCoulombB = NewMol.scaleCoul[j];
      const double r = sqrt(rr_dot);
      const double scalingCoul = scalingCoulombA * scalingCoulombB;
      double resultCoul[2] = {0.0, 0.0};
      CoulombReal(FF.FFParams, chargeA, chargeB, r, scalingCoul, resultCoul);
      tempy += resultCoul[0]; //prefactor merged in the CoulombReal function
    }
  }
  y[ij] = tempy;
  }
}

__device__ __forceinline__ 
double fast_float2double (float a)
{
    unsigned int ia = __float_as_int (a);
    return __hiloint2double ((((ia >> 3) ^ ia) & 0x07ffffff) ^ ia, ia << 29);
}

__global__ void Collapse_Framework_Energy_OVERLAP_PARTIAL(Boxsize Box, Atoms* System, Atoms NewMol, ForceField FF, double* Blocksum, size_t ComponentID, size_t totalAtoms, bool* flag, size_t totalthreads, size_t chainsize, size_t NblockForTrial)
{
  // TEST THE SPEED OF THIS //
  // CHANGED HOW THE ith element of framework positions and y are written/accessed //
  ///////////////////////////////////////////////////////
  //All variables passed here should be device pointers//
  ///////////////////////////////////////////////////////
  extern __shared__ double sdata[]; //shared memory for partial sum//
  int cache_id = threadIdx.x; 
  size_t trial = blockIdx.x/NblockForTrial;
  size_t total_ij = blockIdx.x * blockDim.x + threadIdx.x;
  size_t ij = total_ij - trial * NblockForTrial * blockDim.x;

  size_t ij_within_block = total_ij - blockIdx.x * blockDim.x;

  sdata[ij_within_block] = 0.0; 
  //Initialize Blocksum//
  Blocksum[blockIdx.x] = 0.0; 

  __shared__ bool Blockflag = false;

  if(ij < totalAtoms * chainsize)
  {
  const double OverlapCriteria = FF.FFParams[0];
  const double VDWCutoff = FF.FFParams[1];
  const double CoulCutoff = FF.FFParams[2];
  const bool   noCharges = FF.noCharges;
  // Manually fusing/collapsing the loop //
  //size_t i = ij/Mol.size; size_t j = ij%Mol.size;
  size_t i = ij/chainsize; //ij is the thread id within the trial, just divide by chainsize to get the true i (atom id)
  size_t j = trial*chainsize + ij%chainsize; //+ ij/totalAtoms; // position in NewMol
  //printf("ij: %lu, i: %lu, j: %lu, trial: %lu, totalAtoms: %lu, totalthreads: %lu\n", ij,i,j,k,totalAtoms, totalthreads);
  size_t comp = 0;
  const size_t NumberComp = 2; //Zhao's note: need to change here for multicomponent
  size_t posi = i; size_t totalsize= 0;
  //printf("%lu, %lu\n", System[0].size, System[1].size);
  for(size_t ijk = 0; ijk < NumberComp; ijk++)
  {
    totalsize += System[ijk].size;
    if(posi >= totalsize)
    {
      comp++;
      posi -= System[ijk].size;
    }
  }

  const Atoms Component=System[comp];
  const double scaleA = Component.scale[posi];
  const double chargeA = Component.charge[posi];
  const double scalingCoulombA = Component.scaleCoul[posi];
  const size_t typeA = Component.Type[posi];
  const size_t MoleculeID = Component.MolID[posi];

  double Pos[3] = {Component.x[posi], Component.y[posi], Component.z[posi]};
  double tempy = 0.0;
  //if(j == 6) printf("PAIR CHECK: i: %lu, j: %lu, MoleculeID: %lu, NewMol.MolID: %lu\n", i,j,MoleculeID, NewMol.MolID[0]);
  if(!((MoleculeID == NewMol.MolID[0]) &&(comp == ComponentID)))
  {
    double posvec[3] = {Pos[0] - NewMol.x[j], Pos[1] - NewMol.y[j], Pos[2] - NewMol.z[j]};
    //printf("thread: %lu, i:%lu, j:%lu, comp: %lu, posi: %lu\n", ij,i,j,comp, posi);

    PBC(posvec, Box.Cell, Box.InverseCell, FF.OtherParams);
    const double rr_dot = posvec[0]*posvec[0] + posvec[1]*posvec[1] + posvec[2]*posvec[2];
    if(rr_dot < VDWCutoff)
    {
      double result[2] = {0.0, 0.0};
      const size_t typeB = NewMol.Type[j];
      const double scaleB = NewMol.scale[j];
      const double scaling = scaleA * scaleB;
      const size_t row = typeA*FF.size+typeB;
      const double FFarg[4] = {FF.epsilon[row], FF.sigma[row], FF.z[row], FF.shift[row]};
      VDW(FFarg, rr_dot, scaling, result); 
      if(result[0] > OverlapCriteria){ flag[trial]=true; Blockflag = true; }
      tempy += result[0];
    }

    if (!noCharges && rr_dot < CoulCutoff)
    {
      const double chargeB = NewMol.charge[j];
      const double scalingCoulombB = NewMol.scaleCoul[j];
      const double r = sqrt(rr_dot);
      const double scalingCoul = scalingCoulombA * scalingCoulombB;
      double resultCoul[2] = {0.0, 0.0};
      CoulombReal(FF.FFParams, chargeA, chargeB, r, scalingCoul, resultCoul);
      tempy += resultCoul[0]; //prefactor merged in the CoulombReal function
    }
  }
  sdata[ij_within_block] = tempy;
  }
  __syncthreads();
  //Partial block sum//
  if(!Blockflag)
  {
    int i=blockDim.x / 2;
    while(i != 0) 
    {
      if(cache_id < i) {sdata[cache_id] += sdata[cache_id + i];}
      __syncthreads();
      i /= 2;
    }
    if(cache_id == 0) {Blocksum[blockIdx.x] = sdata[0];}
  }
}

__global__ void Energy_difference_PARTIAL(Boxsize Box, Atoms* System, Atoms Mol, Atoms NewMol, ForceField FF, double* BlockEnergy, double* BlockdUdlambda, size_t ComponentID, size_t totalAtoms, size_t chainsize, size_t Threadsize) // Consider to change this for polyatomic
{
  ///////////////////////////////////////////////////////
  //All variables passed here should be device pointers//
  ///////////////////////////////////////////////////////
  extern __shared__ double2 sdata[]; //shared memory for partial sum, energy + dUdlambda//
  int cache_id = threadIdx.x;
  size_t ij = blockIdx.x * blockDim.x + threadIdx.x;

  size_t ij_within_block = ij - blockIdx.x * blockDim.x;

  sdata[ij_within_block].x = 0.0; sdata[ij_within_block].y = 0.0;
  //Initialize Blocky and BlockdUdlambda//
  BlockEnergy[blockIdx.x] = 0.0; BlockdUdlambda[blockIdx.x] = 0.0;

  __shared__ bool Blockflag = false;

  if(ij < totalAtoms * chainsize)
  {
  BlockEnergy[blockIdx.x] = 0.0; BlockdUdlambda[blockIdx.x] = 0.0;
  const double OverlapCriteria = FF.FFParams[0];
  const double VDWCutoff = FF.FFParams[1];
  const double CoulCutoff = FF.FFParams[2];
  const bool   noCharges = FF.noCharges;
  // Manually fusing/collapsing the loop //
  //size_t i = ij/Mol.size; size_t j = ij%Mol.size;
  size_t i = ij/chainsize; 
  size_t j = ij%chainsize; //+ ij/totalAtoms; // position in Mol and NewMol
  
  size_t comp = 0;
  const size_t NumberComp = 2; //Zhao's note: need to change here for multicomponent
  size_t posi = i; size_t totalsize= 0;
  for(size_t ijk = 0; ijk < NumberComp; ijk++)
  {
    totalsize += System[ijk].size;
    if(posi >= totalsize)
    {
      comp++;
      posi -= System[ijk].size;
    }
  }
  //printf("thread: %lu, comp: %lu, posi: %lu\n", i,comp, posi);

  const Atoms Component=System[comp];
  const double scaleA = Component.scale[posi];
  const double chargeA = Component.charge[posi];
  const double scalingCoulombA = Component.scaleCoul[posi];
  const size_t typeA = Component.Type[posi];
  const size_t MoleculeID = Component.MolID[posi];
  double tempy = 0.0; double tempdU = 0.0;
  if(!((MoleculeID == NewMol.MolID[0]) &&(comp == ComponentID))) //ComponentID: Component ID for the molecule being translated
  {
    ///////////
    //  NEW  //
    ///////////
    double posvec[3] = {Component.x[posi] - NewMol.x[j], Component.y[posi] - NewMol.y[j], Component.z[posi] - NewMol.z[j]};

    PBC(posvec, Box.Cell, Box.InverseCell, FF.OtherParams);
    double rr_dot = posvec[0]*posvec[0] + posvec[1]*posvec[1] + posvec[2]*posvec[2];
    double result[2] = {0.0, 0.0};
    if(rr_dot < FF.FFParams[1])
    {
      const size_t typeB = NewMol.Type[j];
      const double scaleB = NewMol.scale[j];
      const double scaling = scaleA * scaleB;
      const size_t row = typeA*FF.size+typeB;
      const double FFarg[4] = {FF.epsilon[row], FF.sigma[row], FF.z[row], FF.shift[row]};
      VDW(FFarg, rr_dot, scaling, result);
      tempy += result[0];
      tempdU += result[1];
    }
  
    if (!FF.noCharges && rr_dot < FF.FFParams[2])
    {
      const double chargeB = NewMol.charge[j];
      const double scalingCoulombB = NewMol.scaleCoul[j];
      const double r = sqrt(rr_dot);
      const double scalingCoul = scalingCoulombA * scalingCoulombB;
      CoulombReal(FF.FFParams, chargeA, chargeB, r, scalingCoul, result);
      tempy += result[0]; //prefactor merged in the CoulombReal function
    }
    ///////////
    //  OLD  //
    ///////////
    posvec[0] = Component.x[posi] - Mol.x[j]; posvec[1] = Component.y[posi] - Mol.y[j]; posvec[2] = Component.z[posi] - Mol.z[j];
    PBC(posvec, Box.Cell, Box.InverseCell, FF.OtherParams);
    rr_dot = posvec[0]*posvec[0] + posvec[1]*posvec[1] + posvec[2]*posvec[2];
    if(rr_dot < FF.FFParams[1])
    {
      const size_t typeB = Mol.Type[j];
      const double scaleB = Mol.scale[j];
      const double scaling = scaleA * scaleB;
      const size_t row = typeA*FF.size+typeB;
      const double FFarg[4] = {FF.epsilon[row], FF.sigma[row], FF.z[row], FF.shift[row]};
      VDW(FFarg, rr_dot, scaling, result);
      tempy -= result[0];
      tempdU -= result[1];
    }
    if (!FF.noCharges && rr_dot < FF.FFParams[2])
    {
      const double chargeB = Mol.charge[j];
      const double scalingCoulombB = Mol.scaleCoul[j];
      const double r = sqrt(rr_dot);
      const double scalingCoul = scalingCoulombA * scalingCoulombB;
      CoulombReal(FF.FFParams, chargeA, chargeB, r, scalingCoul, result);
      tempy -= result[0]; //prefactor merged in the CoulombReal function
    }
  }
  sdata[ij_within_block].x = tempy; sdata[ij_within_block].y = tempdU;
  }
  __syncthreads();
  //Partial block sum//
  if(!Blockflag)
  {
    int i=blockDim.x / 2;
    while(i != 0)
    {
      if(cache_id < i) 
      {
        sdata[cache_id].x += sdata[cache_id + i].x; //sdata[cache_id].y += sdata[cache_id + i].y;
      }
      __syncthreads();
      i /= 2;
    }
    if(cache_id == 0) 
    { 
      BlockEnergy[blockIdx.x] = sdata[0].x; BlockdUdlambda[blockIdx.x] = sdata[0].y;
    }
  }
}
