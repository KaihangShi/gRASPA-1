#include "hip/hip_runtime.h"
#include "axpy.h"
#include "print_statistics.h"
#include "mc_translation.h"
#include "mc_insertion_deletion.h"
//#include "write_lmp_movie.h"
#include <numeric>
#include <cmath>
#include <algorithm>
#include <filesystem>
/*void CreateMolecules_Before_Simulation(Components& SystemComponents, Boxsize Box, Atoms* System, Atoms* d_a, Atoms Mol, Atoms NewMol, ForceField FF, RandomNumber Random, WidomStruct Widom, Units Constants, bool DualPrecision, std::vector<size_t>& NumberOfCreateMolecules)
{
  double running_energy = 0.0;
  // Create Molecules in the Box Before the Simulation //
  size_t CreateFailCount = 0; size_t Created = 0;
  for(size_t comp = 1; comp < SystemComponents.Total_Components; comp++)
  {
    CreateFailCount = 0; Created = 0;
    while(NumberOfCreateMolecules[comp] > 0)
    {
      printf("Creating %zu Molecule for Component %zu; There are %zu Molecules of that component in the System\n", Created, comp, SystemComponents.NumberOfMolecule_for_Component[comp]);
      size_t OldVal = SystemComponents.NumberOfMolecule_for_Component[comp];
      //running_energy += Insertion(Box, SystemComponents, System, d_a, Mol, NewMol, FF, Random, Widom, Created, comp, DualPrecision);
      running_energy += CreateMolecule(Box, SystemComponents, System, d_a, Mol, NewMol, FF, Random, Widom, Created, comp, DualPrecision);
      if(SystemComponents.NumberOfMolecule_for_Component[comp] == OldVal)
      {CreateFailCount ++;} else {NumberOfCreateMolecules[comp] --; Created ++;}
      if(CreateFailCount > 10) throw std::runtime_error("Bad Insertions When Creating Molecules!");
    }
  }
}*/
double Run_Simulation(int Cycles, Components& SystemComponents, Boxsize Box, Atoms* System, Atoms* d_a, Atoms Mol, Atoms NewMol, ForceField FF, RandomNumber Random, WidomStruct Widom, Units Constants, double init_energy, bool DualPrecision, std::vector<size_t>& NumberOfCreateMolecules, bool CreateMolecules)
{
  
  double running_energy = 0.0;

  size_t WidomCount = 0;

  bool DEBUG = false;
  size_t transCount=0;

   // Create Molecules in the Box Before the Simulation //
  if(CreateMolecules){
  size_t CreateFailCount = 0; size_t Created = 0; size_t SelectedMol = 0;
  for(size_t comp = 1; comp < SystemComponents.Total_Components; comp++)
  {
    CreateFailCount = 0;
    while(NumberOfCreateMolecules[comp] > 0)
    {
      printf("Creating %zu Molecule for Component %zu; There are %zu Molecules of that component in the System\n", Created, comp, SystemComponents.NumberOfMolecule_for_Component[comp]);
      SelectedMol = Created; if(Created > 0) SelectedMol = Created - 1; //Zhao's note: this is a little confusing, but when number of molecule for that species = 0 or 1, the chosen molecule is zero. This is creating from zero loading, need to change in the future, when we read from restart file//
      size_t OldVal = SystemComponents.NumberOfMolecule_for_Component[comp];
      //running_energy += Insertion(Box, SystemComponents, System, d_a, Mol, NewMol, FF, Random, Widom, SelectedMol, comp, DualPrecision);
      running_energy += CreateMolecule(Box, SystemComponents, System, d_a, Mol, NewMol, FF, Random, Widom, SelectedMol, comp, DualPrecision);
      if(SystemComponents.NumberOfMolecule_for_Component[comp] == OldVal)
      {CreateFailCount ++;} else {NumberOfCreateMolecules[comp] --; Created ++;}
      if(CreateFailCount > 10) throw std::runtime_error("Bad Insertions When Creating Molecules!");
    }
  }
  return running_energy;
  }
  

  printf("There are %zu Molecules, %zu Frameworks\n",SystemComponents.TotalNumberOfMolecules, SystemComponents.NumberOfFrameworks);

  for(size_t i = 0; i < Cycles; i++)
  {
    //Randomly Select an Adsorbate Molecule and determine its Component: MoleculeID --> Component
    //if((SystemComponents.TotalNumberOfMolecules - SystemComponents.NumberOfFrameworks) == 0)
    //  continue;
    size_t SelectedMolecule = (size_t) (get_random_from_zero_to_one()*(SystemComponents.TotalNumberOfMolecules-SystemComponents.NumberOfFrameworks));
    size_t comp = SystemComponents.NumberOfFrameworks; // When selecting components, skip the component 0 (because it is the framework)
    size_t SelectedMolInComponent = SelectedMolecule; size_t totalsize= 0;
    for(size_t ijk = SystemComponents.NumberOfFrameworks; ijk < SystemComponents.Total_Components; ijk++) //Assuming Framework atoms are the top in the Atoms array
    {
      if(SelectedMolInComponent == 0) break;
      totalsize += SystemComponents.NumberOfMolecule_for_Component[ijk];
      if(SelectedMolInComponent >= totalsize)
      {
        comp++;
        SelectedMolInComponent -= SystemComponents.NumberOfMolecule_for_Component[ijk];
      }
    }

    //printf("Selected Comp: %zu, SelectedMol: %zu, Num: %zu\n", comp, SelectedMolInComponent, SystemComponents.NumberOfMolecule_for_Component[comp]);
    if(SystemComponents.NumberOfMolecule_for_Component[comp] == 0){ //no molecule in the system for this species
      //printf("Doing insertion since there is no molecule for this species; SelectedMol: %zu, comp: %zu\n", SelectedMolInComponent, comp);
      running_energy += Insertion(Box, SystemComponents, System, d_a, Mol, NewMol, FF, Random, Widom, SelectedMolInComponent, comp, DualPrecision);
      continue;
    }

    double RANDOMNUMBER = get_random_from_zero_to_one();
    if(RANDOMNUMBER < SystemComponents.Moves[comp].TranslationProb)
    {
      transCount++;
      //PERFORM TRANSLATION MOVE//
      running_energy += Translation_Move(Box, SystemComponents, System, d_a, Mol, NewMol, Widom, FF, Random, SelectedMolInComponent, comp);
      if(DEBUG){printf("After Translation: running energy: %.10f\n", running_energy);}
    }
    else if(RANDOMNUMBER < SystemComponents.Moves[comp].RotationProb) //Rotation
    {
      //PERFORM ROTATION MOVE, a test//
      running_energy += Rotation_Move(Box, SystemComponents, System, d_a, Mol, NewMol, Widom, FF, Random, SelectedMolInComponent, comp);
      //printf("After Translation: running energy: %.10f\n", running_energy);
    }
    else if(RANDOMNUMBER < SystemComponents.Moves[comp].WidomProb)
    {
      WidomCount ++;
      //printf("Performing Widom\n");
      size_t SelectedTrial=0; bool SuccessConstruction = false; double energy = 0.0; double StoredR = 0.0;
      double Rosenbluth=Widom_Move_FirstBead_PARTIAL(Box, SystemComponents, System, d_a, NewMol, FF, Random, Widom, SelectedMolInComponent, comp, Insertion, false, false, StoredR, &SelectedTrial, &SuccessConstruction, &energy, false);
      //double Rosenbluth=Widom_Move_FirstBead(Box, SystemComponents, System, d_a, NewMol, FF, Random, Widom, SelectedMolInComponent, comp, Insertion, false, false, StoredR, &SelectedTrial, &SuccessConstruction, &energy, false); //first false: Reinsertion? second false: Retrace? third false is for using Dual-Precision. For Widom Insertion, don't use it.//
      if(SystemComponents.Moleculesize[comp] > 1 && Rosenbluth > 1e-150)
      {
        size_t SelectedFirstBeadTrial = SelectedTrial; 
        Rosenbluth*=Widom_Move_Chain_PARTIAL(Box, SystemComponents, System, d_a, Mol, NewMol, FF, Random, Widom, SelectedMolInComponent, comp, Insertion, false, &SelectedTrial, &SuccessConstruction, &energy, SelectedFirstBeadTrial, false); //false is for using Dual-Precision. For Widom Insertion, don't use it.//
      }
      //Assume 5 blocks
      size_t BlockIDX = i/(Cycles/SystemComponents.Moves[comp].NumberOfBlocks); //printf("BlockIDX=%zu\n", BlockIDX);
      Widom.Rosenbluth[BlockIDX]+= Rosenbluth;
      Widom.RosenbluthSquared[BlockIDX]+= Rosenbluth*Rosenbluth;
      Widom.RosenbluthCount[BlockIDX]++;
    }
    else if(RANDOMNUMBER < SystemComponents.Moves[comp].ReinsertionProb)
    {
      if(DEBUG) printf("Before Reinsertion, energy: %.10f\n", running_energy);
      running_energy += Reinsertion(Box, SystemComponents, System, d_a, Mol, NewMol, FF, Random, Widom, SelectedMolInComponent, comp, DualPrecision);
    }
    else
    {
      // DO GCMC INSERTION //
      //if((DEBUG) && (i < 20)){
      if(get_random_from_zero_to_one() < 0.5){ //0.5){
        //printf("Doing insertion SelectedMol: %zu, comp: %zu\n", SelectedMolInComponent, comp);
        running_energy += Insertion(Box, SystemComponents, System, d_a, Mol, NewMol, FF, Random, Widom, SelectedMolInComponent, comp, DualPrecision);}
      else{
        if(DEBUG){printf("Cycle: %zu, DOING DELETION\n", i);}
        running_energy += Deletion(Box, SystemComponents, System, d_a, Mol, NewMol, FF, Random, Widom, SelectedMolInComponent, comp, DualPrecision);}
    }
    if(i%500==0 &&(SystemComponents.Moves[comp].TranslationTotal > 0))
    {
      printf("i: %zu\n", i);
      Update_Max_Translation(FF, SystemComponents.Moves[comp]);
    }
    if(DEBUG)
    {
      printf("After %zu MOVE: Sum energies\n", i);
      double* xxx; xxx = (double*) malloc(sizeof(double)*2);
      double* device_xxx = CUDA_copy_allocate_double_array(xxx, 2);
      one_thread_GPU_test<<<1,1>>>(Box, d_a, FF, device_xxx); hipMemcpy(xxx, device_xxx, sizeof(double), hipMemcpyDeviceToHost);
      printf("Current Total Energy (1 thread GPU): %.10f, running total: %.10f\n", xxx[0], init_energy+running_energy);
      hipDeviceSynchronize();
      if(abs(xxx[0] - (init_energy+running_energy)) > 0.1) //means that there is an energy drift
      {
        printf("THere is an energy drift at cycle %zu\n", i);
      }
      hipFree(device_xxx);
    }
  }
  //print statistics
  for(size_t comp = SystemComponents.NumberOfFrameworks; comp < SystemComponents.Total_Components; comp++)
  {
    Print_Translation_Statistics(SystemComponents.Moves[comp], FF);
    Print_Rotation_Statistics(SystemComponents.Moves[comp], FF);
    Print_Widom_Statistics(Widom, SystemComponents.Moves[comp], SystemComponents.Beta, Constants.energy_to_kelvin);
    Print_Swap_Statistics(Widom, SystemComponents.Moves[comp]);
    //printf("TransCount: %zu\n", transCount);
    //printf("total-deltaU: %.10f\n", running_energy);
  }
  return running_energy;
}
