#include "GPU_alloc.cuh"
size_t* CUDA_copy_allocate_size_t_array(size_t* x, size_t N)
{
  size_t* device_x;
  hipMalloc(&device_x, N * sizeof(size_t)); checkCUDAError("Error allocating Malloc");
  hipMemcpy(device_x, x, N * sizeof(size_t), hipMemcpyHostToDevice); checkCUDAError("size_t Error Memcpy");
  return device_x;
}

int* CUDA_copy_allocate_int_array(int* x, size_t N)
{
  int* device_x;
  hipMalloc(&device_x, N * sizeof(int)); checkCUDAError("Error allocating Malloc");
  hipMemcpy(device_x, x, N * sizeof(int), hipMemcpyHostToDevice); checkCUDAError("int Error Memcpy");
  return device_x;
}

double* CUDA_copy_allocate_double_array(double* x, size_t N)
{
  double* device_x;
  hipMalloc(&device_x, N * sizeof(double)); checkCUDAError("Error allocating Malloc");
  hipMemcpy(device_x, x, N * sizeof(double), hipMemcpyHostToDevice); checkCUDAError("double Error Memcpy");
  return device_x;
}
double* CUDA_allocate_double_array(size_t N)
{
  double* device_x;
  hipMalloc(&device_x, N * sizeof(double)); checkCUDAError("Error allocating Malloc");
  return device_x;
}

void CUDA_copy_double_array(double* x, double **device_x, size_t N)
{
  hipMemcpy(*device_x, x, N * sizeof(double), hipMemcpyHostToDevice); checkCUDAError("Just double Error Memcpy");
}

